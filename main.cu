#include "hip/hip_runtime.h"
﻿// GPU CUDA Computations.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include <iostream>
#include <math.h>
#include <string.h>
#include <chrono>
#include "CUDA_Header.cuh"
#include "hip/hip_runtime.h"
#include "Adding.h"

int main(void)
{
    const int N = 1 << 15; //1024 operations is the when they are approximately equal on my machine (1 << 10)

    float* x;
    float* y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    int iterations = 10000;
    auto start = std::chrono::high_resolution_clock::now();
    // Run kernel on 1M elements on the CPU
    int blockS = 512;
    int blockN = (N + blockS - 1) / blockS;
    for (int ite = 0; ite < iterations; ite++)
    {
        addKer<<<blockN, blockS>>> (N, x, y);
    }
    //addKerCpu(N, x, y);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    std::cout << "CUDA Multithreaded: " << duration.count()/iterations << "ns\n";
    hipDeviceSynchronize();


    // Free memory
    hipFree(x);
    hipFree(y);

    float xn[N];
    float yn[N];
    
    for (int i = 0; i < N; i++) {
        xn[i] = 1.0f;
        yn[i] = 2.0f;
    }

    auto start2 = std::chrono::high_resolution_clock::now();
    for (int iters = 0; iters < iterations; iters++)
    {
        for (int i = 0; i < N; i++) {
            CPU(xn, yn, i);
        }
    }
    
    auto stop2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::nanoseconds>(stop2 - start2);
    std::cout << "Single CPU Thread: " << duration2.count() / iterations << "ns";
    std::string c;
    std::cin >> c;

    return 0;
}